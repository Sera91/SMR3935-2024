/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 */

// -----------------------------------------------------------------------------
// Transpose
//
// This file contains both device and host code for transposing a floating-point
// matrix.  It performs several transpose kernels, which incrementally improve
// performance through coalescing, removing shared memory bank conflicts, and
// eliminating partition camping.  Several of the kernels perform a copy, used
// to represent the best case performance that a transpose can achieve.
//
// Please see the whitepaper in the docs folder of the transpose project for a
// detailed description of this performance study.
// -----------------------------------------------------------------------------


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
//#include <cooperative_groups.h>

#define checkCudaErrors( func_call ) func_call

// Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// using TILE_DIM x BLOCK_ROWS threads, so that each thread transposes
// TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an integral multiple of
// BLOCK_ROWS

#define TILE_DIM 16
#define BLOCK_ROWS 16

// This sample assumes that MATRIX_SIZE_X = MATRIX_SIZE_Y
int MATRIX_SIZE_X = 1024;
int MATRIX_SIZE_Y = 1024;
int MUL_FACTOR = TILE_DIM;

#define FLOOR(a, b) (a - (a % b))

// Compute the tile size necessary to illustrate performance cases for SM20+
// hardware
int MAX_TILES = (FLOOR(MATRIX_SIZE_X, 512) * FLOOR(MATRIX_SIZE_Y, 512)) /
                (TILE_DIM * TILE_DIM);

// Number of repetitions used for timing.  Two sets of repetitions are
// performed: 1) over kernel launches and 2) inside the kernel over just the
// loads and stores

#define NUM_REPS 100

// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void copy(float *odata, float *idata, int width, int height) {
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index = xIndex + width * yIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index + i * width] = idata[index + i * width];
  }
}

__global__ void copySharedMem(float *odata, float *idata, int width,
                              int height) {
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index = xIndex + width * yIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (xIndex < width && yIndex < height) {
      tile[threadIdx.y][threadIdx.x] = idata[index];
    }
  }

  __syncthreads();

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (xIndex < height && yIndex < width) {
      odata[index] = tile[threadIdx.y][threadIdx.x];
    }
  }
}

// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float *idata, int width,
                               int height) {
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i] = idata[index_in + i * width];
  }
}

// coalesced transpose (with bank conflicts)

__global__ void transposeCoalesced(float *odata, float *idata, int width,
                                   int height) {
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  __syncthreads();

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
  }
}

// Coalesced transpose with no bank conflicts

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width,
                                         int height) {
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  __syncthreads();

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
  }
}

// ---------------------
// host utility routines
// ---------------------

void computeTransposeCPU(float *gold, float *idata, const int size_x,
                         const int size_y) {
  for (int y = 0; y < size_y; ++y) {
    for (int x = 0; x < size_x; ++x) {
      gold[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

//////////////////////////////////////////////////////////////////////////////
//! Compare two arrays of arbitrary type
//! @return  true if \a reference and \a data are identical, otherwise false
//! @param reference  timer_interface to the reference data / gold image
//! @param data       handle to the computed data
//! @param len        number of elements in reference and data
//! @param epsilon    epsilon to use for the comparison
//////////////////////////////////////////////////////////////////////////////

inline bool compareData(const float *reference, const float *data,
                        const unsigned int len, const float epsilon,
                        const float threshold) {
  assert(epsilon >= 0);

  bool result = true;
  unsigned int error_count = 0;

  for (unsigned int i = 0; i < len; ++i) {
    float diff = static_cast<float>(reference[i]) - static_cast<float>(data[i]);
    bool comp = (diff <= epsilon) && (diff >= -epsilon);
    result &= comp;

    error_count += !comp;
  }

  if (threshold == 0.0f) {
    return (result) ? true : false;
  } else {
    if (error_count) {
      printf("%4.2f(%%) of bytes mismatched (count=%d)\n",
             static_cast<float>(error_count) * 100 / static_cast<float>(len),
             error_count);
    }

    return (len * threshold > error_count) ? true : false;
  }
}

int main(int argc, char **argv)
{
  if (argc != 2) {
    printf("Transpose: Must specify a matrix dimension\n");
    return 1;
  }

  // Matrix dimensions
  int size_x = 512;
  size_x = atoi(argv[1]);
  int size_y = size_x;

  float total_tiles = (float)MAX_TILES;

  if (size_x % TILE_DIM != 0 || size_y % TILE_DIM != 0) {
    printf("Matrix size must be integral multiple of tile size %d\nExiting...\n\n", TILE_DIM);
    exit(EXIT_FAILURE);
  }

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int);
  const char *kernelName;

  // execution configuration parameters
  dim3 grid(size_x / TILE_DIM, size_y / TILE_DIM);
  dim3 threads(TILE_DIM, BLOCK_ROWS);

  if (grid.x < 1 || grid.y < 1) {
    printf("Grid size computation incorrect in test\nExiting...\n\n");
    exit(EXIT_FAILURE);
  }

  // CUDA events
  hipEvent_t start, stop;

  // size of memory required to store the matrix
  size_t mem_size = static_cast<size_t>(sizeof(float) * size_x * size_y);

  // allocate host memory
  float *h_idata = (float *)malloc(mem_size);
  float *h_odata = (float *)malloc(mem_size);
  float *transposeGold = (float *)malloc(mem_size);
  float *gold;

  // allocate device memory
  float *d_idata, *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
  checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));

  // initialize host data
  for (int i = 0; i < (size_x * size_y); ++i) {
    h_idata[i] = (float)i;
  }

  // copy host data to device
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // Compute reference transpose solution
  computeTransposeCPU(transposeGold, h_idata, size_x, size_y);

  // print out common data for all kernels
  printf(
      "\nMatrix size: %dx%d (%dx%d tiles), tile size: %dx%d, block size: "
      "%dx%d\n\n",
      size_x, size_y, size_x / TILE_DIM, size_y / TILE_DIM, TILE_DIM, TILE_DIM,
      TILE_DIM, BLOCK_ROWS);

  // initialize events
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  //
  // loop over different kernels
  //

  bool success = true;

  for (int k = 0; k < 5; k++) {
    // set kernel pointer
    switch (k) {
      case 0:
        kernel = &copy;
        kernelName = "simple copy       ";
        break;

      case 1:
        kernel = &copySharedMem;
        kernelName = "shared memory copy";
        break;

      case 2:
        kernel = &transposeNaive;
        kernelName = "naive             ";
        break;

      case 3:
        kernel = &transposeCoalesced;
        kernelName = "coalesced         ";
        break;

      case 4:
        kernel = &transposeNoBankConflicts;
        kernelName = "optimized         ";
        break;

      default:
        break;
    }

    // set reference solution
    if (kernel == &copy || kernel == &copySharedMem) {
      gold = h_idata;
    } else {
      gold = transposeGold;
    }

    // Clear error status
    checkCudaErrors(hipGetLastError());

    // warmup to avoid timing startup
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y);

    // take measurements for loop over kernel launches
    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; i++) {
      kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y);
      // Ensure no launch failure
      checkCudaErrors(hipGetLastError());
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float kernelTime;
    checkCudaErrors(hipEventElapsedTime(&kernelTime, start, stop));

    checkCudaErrors(
        hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));

    bool res = compareData(gold, h_odata, size_x * size_y, 0.01f, 0.0f);

    if (res == false) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = false;
    }

    // report effective bandwidths
    float kernelBandwidth = 2.0f * 1000.0f * mem_size / (1024 * 1024 * 1024) /
                            (kernelTime / NUM_REPS);
    printf(
        "transpose %s, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u fp32 "
        "elements, NumDevsUsed = %u, Workgroup = %u\n",
        kernelName, kernelBandwidth, kernelTime / NUM_REPS, (size_x * size_y),
        1, TILE_DIM * BLOCK_ROWS);
  }

  // cleanup
  free(h_idata);
  free(h_odata);
  free(transposeGold);
  hipFree(d_idata);
  hipFree(d_odata);

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  if (!success) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
