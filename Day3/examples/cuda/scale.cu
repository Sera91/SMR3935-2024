/**
 * Vector scale: x <= x * k
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
//#include <cuda_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Multiplies all the elements of vector x by a scalar value
 */
__global__ void scale(float *x, const float k, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    x[i] = x[i] * k;
  }
}

/**
 * Host main routine
 */
int main(void) {

  float k = 10.0;

  // Print the vector length to be used, and compute its size
  int numElements = 50000;
  size_t size = numElements * sizeof(float);
  printf("Vector scaling of %d elements: k = %f\n", numElements, k);

  // Allocate the host input vector x
  float *h_x = (float *)malloc(size);

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_x[i] = i;
  }

  hipError_t status;

  // Allocate the device input vector x
  float *d_x = NULL;
  status = hipMalloc((void **)&d_x, size);
  if (status != hipSuccess) {
    printf("error cudaMalloc: error code %d: %s\n", status, hipGetErrorString(status));
  }

  // Copy the host input vector x in host memory to the device input
  // vectors in device memory
  
  status = hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
  if (status != hipSuccess) {
    printf("error cudaMemcpy: error code %d: %s\n", status, hipGetErrorString(status));
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  scale<<< blocksPerGrid, threadsPerBlock >>>(d_x, k, numElements);
  status = hipGetLastError();
  if (status != hipSuccess) {
    printf("error kernel launch: error code %d: %s\n", status, hipGetErrorString(status));
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);

  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_x[i]  - k * i) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  hipFree(d_x);

  // Free host memory
  free(h_x);

  printf("Done\n");
  return 0;
}
